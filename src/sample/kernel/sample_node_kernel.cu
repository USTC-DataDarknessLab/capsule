#include <stdio.h>
#include <stdlib.h>
#include <cassert>
#include <chrono>
#include <numeric>
#include <exception>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_mtgp32_host.h>
#include <rocrand/rocrand_mtgp32_11213.h>

__global__ void sample_full_kernel(
                            int* outputSRC,
                            int* outputDST,
                            const int* graphEdge,
                            const int* boundList,
                            const int* trainNode,
                            int nodeNUM) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for(int i = idx ; i < nodeNUM ; i += blockDim.x) {
        int writeIdx = i * 25;
        int id = trainNode[i];
        int idxStart = boundList[id];
        int idxEnd = boundList[id+1];
        for (int l = 0 ; l < (idxEnd - idxStart) ; l++) {
            outputSRC[writeIdx] = graphEdge[idxStart + l];
            outputDST[writeIdx++] = id;
        }

    }    
}

__global__ void sample1Hop(
                        int* outputSRC1,
                        int* outputDST1, 
                        const int* graphEdge,
                        const int* boundList,
                        const int* trainNode,
                        int sampleNUM1,
                        int nodeNUM,
                        unsigned long long seed
                            ) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandStateXORWOW_t state;
    hiprand_init(seed+idx,0,0,&state);
    unsigned int random_value = 0;
    int blockSize = sampleNUM1;
    for(int i = idx ; i < nodeNUM ; i += blockDim.x) {
        int writeIdx = i * blockSize;
        int id = trainNode[i];
        int idxStart = boundList[id];
        int idxEnd = boundList[id+1];
        int neirNUM = idxEnd - idxStart;
        for (int l = 0 ; l < neirNUM ; l++) { 
            random_value = hiprand(&state) % neirNUM;
            outputSRC1[writeIdx] = graphEdge[idxStart + random_value];
            outputDST1[writeIdx++] = id;
        }
        for (int l = neirNUM; l < sampleNUM1 ; l++) {
            outputSRC1[writeIdx] = 0;
            outputDST1[writeIdx++] = id;
        }
    }
}

__global__ void sample2Hop(
                        int* outputSRC1,
                        int* outputDST1,
                        int* outputSRC2,
                        int* outputDST2,
                        const int* graphEdge,
                        const int* boundList,
                        const int* trainNode,
                        int sampleNUM1,
                        int sampleNUM2,
                        int nodeNUM,
                        unsigned long long seed
                            ) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandStateXORWOW_t state;
    hiprand_init(seed+idx,0,0,&state);
    unsigned int random_value = 0;
    for(int i = idx ; i < nodeNUM ; i += blockDim.x) {
        int writeIdx = i * sampleNUM1;
        int id = trainNode[i];
        int idxStart = boundList[id];
        int idxEnd = boundList[id+1];
        int neirNUM = idxEnd - idxStart;
        for (int l = 0 ; l < neirNUM && l < sampleNUM1 ; l++) {
            random_value = hiprand(&state) % neirNUM;
            outputSRC1[writeIdx] = graphEdge[idxStart + random_value];
            outputDST1[writeIdx++] = id;
        }
        for (int l = neirNUM; l < sampleNUM1 ; l++) {
            outputSRC1[writeIdx] = -1;
            outputDST1[writeIdx++] = id;
        }

        // hop-2
        for (int l1 = 0 ; l1 < sampleNUM1 ; l1++) {
            // 二层采样id
            int l2_id = outputSRC1[i * sampleNUM1 + l1];
            if (l2_id > 0) {
                int l2_writeIdx = i*sampleNUM1*sampleNUM2 + l1*sampleNUM2;
                int l2_idStart = boundList[l2_id];
                int l2_idEnd = boundList[l2_id+1];
                int l2_neirNUM = l2_idEnd - l2_idStart;
                for (int l = 0 ; l < l2_neirNUM && l < sampleNUM2 ; l++) {
                    random_value = hiprand(&state) % l2_neirNUM;
                    outputSRC2[l2_writeIdx] = graphEdge[l2_idStart + random_value];
                    outputDST2[l2_writeIdx++] = l2_id;
                }
                for (int l = l2_neirNUM; l < sampleNUM2 ; l++) {
                    outputSRC2[l2_writeIdx] = -1;
                    outputDST2[l2_writeIdx++] = l2_id;
                }
            } else {
                int l2_writeIdx = i*sampleNUM1*sampleNUM2 + l1*sampleNUM2;
                for (int l = 0 ; l < sampleNUM2 ; l++) {
                    outputSRC2[l2_writeIdx] = -1;
                    outputDST2[l2_writeIdx++] = l2_id;
                }
            }
        }
    }
}

__global__ void sample3Hop(
                        int* outputSRC1,int* outputDST1,
                        int* outputSRC2,int* outputDST2,
                        int* outputSRC3,int* outputDST3,
                        const int* graphEdge,
                        const int* boundList,
                        const int* trainNode,
                        int sampleNUM1,int sampleNUM2,int sampleNUM3,
                        int nodeNUM,
                        unsigned long long seed
                            ) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandStateXORWOW_t state;
    hiprand_init(seed+idx,0,0,&state);
    unsigned int random_value = 0;
    for(int i = idx ; i < nodeNUM ; i += blockDim.x) {
        int writeIdx = i * sampleNUM1;
        int id = trainNode[i];
        int idxStart = boundList[id];
        int idxEnd = boundList[id+1];
        int neirNUM = idxEnd - idxStart;
        for (int l = 0 ; l < neirNUM && l < sampleNUM1 ; l++) {
            random_value = hiprand(&state) % neirNUM;
            outputSRC1[writeIdx] = graphEdge[idxStart + random_value];
            outputDST1[writeIdx++] = id;
        }
        for (int l = neirNUM; l < sampleNUM1 ; l++) {
            outputSRC1[writeIdx] = -1;
            outputDST1[writeIdx++] = id;
        }

        // hop-2
        for (int l1 = 0 ; l1 < sampleNUM1 ; l1++) {
            // 二层采样id
            int l2_id = outputSRC1[i * sampleNUM1 + l1];
            if (l2_id > 0) {
                int l2_writeIdx = i*sampleNUM1*sampleNUM2 + l1*sampleNUM2;
                int l2_idStart = boundList[l2_id];
                int l2_idEnd = boundList[l2_id+1];
                int l2_neirNUM = l2_idEnd - l2_idStart;
                for (int l = 0 ; l < l2_neirNUM && l < sampleNUM2 ; l++) {
                    random_value = hiprand(&state) % l2_neirNUM; 
                    outputSRC2[l2_writeIdx] = graphEdge[l2_idStart + random_value];
                    outputDST2[l2_writeIdx++] = l2_id;
                }
                for (int l = l2_neirNUM; l < sampleNUM2 ; l++) {
                    outputSRC2[l2_writeIdx] = -1;
                    outputDST2[l2_writeIdx++] = l2_id;
                }
            } else {
                int l2_writeIdx = i*sampleNUM1*sampleNUM2 + l1*sampleNUM2;
                for (int l = 0 ; l < sampleNUM2 ; l++) {
                    outputSRC2[l2_writeIdx] = -1;
                    outputDST2[l2_writeIdx++] = l2_id;
                }
            }
            
        }

        for (int l2 = 0 ; l2 < sampleNUM2 ; l2++) {
            int l3_id = outputSRC2[i * sampleNUM2 + l2];
            if (l3_id > 0) {
                int l3_writeIdx = i*sampleNUM2*sampleNUM3 + l2*sampleNUM3;
                int l3_idStart = boundList[l3_id];
                int l3_idEnd = boundList[l3_id+1];
                int l3_neirNUM = l3_idEnd - l3_idStart;
                for (int l = 0 ; l < l3_neirNUM && l < sampleNUM3 ; l++) {
                    random_value = hiprand(&state) % l3_neirNUM; 
                    outputSRC3[l3_writeIdx] = graphEdge[l3_idStart + random_value];
                    outputDST3[l3_writeIdx++] = l3_id;
                }
                for (int l = l3_neirNUM; l < sampleNUM3 ; l++) {
                    outputSRC3[l3_writeIdx] = -1;
                    outputDST3[l3_writeIdx++] = l3_id;
                }
            } else {
                int l3_writeIdx = i*sampleNUM2*sampleNUM3 + l2*sampleNUM3;
                for (int l = 0 ; l < sampleNUM2 ; l++) {
                    outputSRC3[l3_writeIdx] = -1;
                    outputDST3[l3_writeIdx++] = l3_id;
                }
            }
        }
    }
}

void launch_sample_full(int* outputSRC1,
                 int* outputDST1,
                 const int* graphEdge,
                 const int* boundList,
                 const int* trainNode,
                 int n,
                 const int gpuDeviceIndex
                 ) {
    dim3 grid((n + 1023) / 1024);
    dim3 block(1024);
    
    /* 指定使用的GPU序号 [0,torch.cuda.device_count()) */
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        //printf("No GPU devices found.\n");
        return;
    }
    else if(gpuDeviceIndex >= deviceCount || gpuDeviceIndex < 0){
        //printf("Wrong GPU Device Index:%d , Select Default Device Index:0 cuda:0.\n",gpuDeviceIndex);
        hipSetDevice(0);
    }
    else{
        //printf("Select GPU Device Index:%d , Please Prepare Pytorch Data tensor.to(device='cuda:%d')\n",gpuDeviceIndex,gpuDeviceIndex);
        hipSetDevice(gpuDeviceIndex);
    }

    sample_full_kernel<<<grid, block>>>(outputSRC1, outputDST1, graphEdge, boundList, trainNode, n);
}

void launch_sample_1hop(int* outputSRC1,
                        int* outputDST1, 
                        const int* graphEdge,
                        const int* boundList,
                        const int* trainNode,
                        int sampleNUM1,
                        int nodeNUM,
                        const int gpuDeviceIndex
                        ) {
    dim3 grid((nodeNUM + 1023) / 1024);
    dim3 block(1024);
    unsigned long long seed = std::chrono::system_clock::now().time_since_epoch().count();
    
    /* 指定使用的GPU序号 [0,torch.cuda.device_count()) */
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        //printf("No GPU devices found.\n");
        return;
    }
    else if(gpuDeviceIndex >= deviceCount || gpuDeviceIndex < 0){
        //printf("Wrong GPU Device Index:%d , Select Default Device Index:0 cuda:0.\n",gpuDeviceIndex);
        hipSetDevice(0);
    }
    else{
        //printf("Select GPU Device Index:%d , Please Prepare Pytorch Data tensor.to(device='cuda:%d')\n",gpuDeviceIndex,gpuDeviceIndex);
        hipSetDevice(gpuDeviceIndex);
    }

    

    //auto t_beg = std::chrono::high_resolution_clock::now();
    sample1Hop<<<grid, block>>>(
        outputSRC1,outputDST1,graphEdge,
        boundList,trainNode,sampleNUM1,
        nodeNUM,seed);
    //auto t_end = std::chrono::high_resolution_clock::now();
    //printf("sample1Hop time in function`launch_sample_1hop` : %lf ms\n",std::chrono::duration<double, std::milli>(t_end-t_beg).count());
}

void launch_sample_2hop(int* outputSRC1,
                        int* outputDST1,
                        int* outputSRC2,
                        int* outputDST2,
                        const int* graphEdge,
                        const int* boundList,
                        const int* trainNode,
                        int sampleNUM1,
                        int sampleNUM2,
                        int nodeNUM,
                        const int gpuDeviceIndex
                        ) {
    dim3 grid((nodeNUM + 1023) / 1024);
    dim3 block(1024);
    unsigned long long seed = std::chrono::system_clock::now().time_since_epoch().count();

    /* 指定使用的GPU序号 [0,torch.cuda.device_count()) */
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        //printf("No GPU devices found.\n");
        return;
    }
    else if(gpuDeviceIndex >= deviceCount || gpuDeviceIndex < 0){
        //printf("Wrong GPU Device Index:%d , Select Default Device Index:0 cuda:0.\n",gpuDeviceIndex);
        hipSetDevice(0);
    }
    else{
        //printf("Select GPU Device Index:%d , Please Prepare Pytorch Data tensor.to(device='cuda:%d')\n",gpuDeviceIndex,gpuDeviceIndex);
        hipSetDevice(gpuDeviceIndex);
    }

    //auto t_beg = std::chrono::high_resolution_clock::now();
    sample2Hop<<<grid, block>>>(
        outputSRC1,outputDST1,outputSRC2,
        outputDST2,graphEdge,boundList,
        trainNode,sampleNUM1,sampleNUM2,nodeNUM,seed);
    //auto t_end = std::chrono::high_resolution_clock::now();
    //printf("sample2Hop time in function`launch_sample_2hop` : %lf ms\n",std::chrono::duration<double, std::milli>(t_end-t_beg).count());
}

void launch_sample_3hop(int* outputSRC1,int* outputDST1,
                        int* outputSRC2,int* outputDST2,
                        int* outputSRC3,int* outputDST3,
                        const int* graphEdge,
                        const int* boundList,
                        const int* trainNode,
                        int sampleNUM1,int sampleNUM2,int sampleNUM3,
                        int nodeNUM,
                        const int gpuDeviceIndex
                        ) {
    dim3 grid((nodeNUM + 1023) / 1024);
    dim3 block(1024);
    unsigned long long seed = std::chrono::system_clock::now().time_since_epoch().count();

    /* 指定使用的GPU序号 [0,torch.cuda.device_count()) */
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        //printf("No GPU devices found.\n");
        return;
    }
    else if(gpuDeviceIndex >= deviceCount || gpuDeviceIndex < 0){
        //printf("Wrong GPU Device Index:%d , Select Default Device Index:0 cuda:0.\n",gpuDeviceIndex);
        hipSetDevice(0);
    }
    else{
        //printf("Select GPU Device Index:%d , Please Prepare Pytorch Data tensor.to(device='cuda:%d')\n",gpuDeviceIndex,gpuDeviceIndex);
        hipSetDevice(gpuDeviceIndex);
    }

    //auto t_beg = std::chrono::high_resolution_clock::now();
    sample3Hop<<<grid, block>>>(
        outputSRC1,outputDST1,outputSRC2,
        outputDST2,outputSRC3,outputDST3,
        graphEdge,boundList,trainNode,
        sampleNUM1,sampleNUM2,sampleNUM3,nodeNUM,seed);
    //auto t_end = std::chrono::high_resolution_clock::now();
    //printf("sample3Hop time in function`launch_sample_3hop` : %lf ms\n",std::chrono::duration<double, std::milli>(t_end-t_beg).count());
}

__global__ void func0(int* cacheData0,
                    int* cacheData1,
                    const int* edges,
                    const int cacheData0Len,
                    const int cacheData1Len,
                    const int edgesLen,
                    const int graphEdgeNUM)
{
    int lastid = -1;
    int endidx = -1;
    int nextidx = -1;
    for(int i = 0;i < edgesLen/2;i++)
    {
        int src = edges[i*2];
        int dst = edges[i*2 + 1];
        if(dst != lastid)
        {
            if(cacheData1Len > dst*2+2){
                endidx = cacheData1[dst*2+1];
                nextidx = cacheData1[dst*2+2];
            }
            else{
                nextidx = graphEdgeNUM;
            }
            lastid = dst;
        }
        
        if(endidx < nextidx)
        {
            if(endidx < cacheData0Len)
                cacheData0[endidx] = src;
            endidx += 1;
        }
    }
}

__global__ void func1(int* cacheData0,
                    int* cacheData1,
                    const int* edges,
                    const int* bound,
                    const int cacheData0Len,
                    const int cacheData1Len,
                    const int edgesLen,
                    const int boundLen,
                    const int graphEdgeNUM)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= (boundLen-1))
        return;
    const int lowerBound = bound[idx];
    const int upperBound = bound[idx+1];
    const int dst = edges[lowerBound+1];
    int endidx = -1;
    int nextidx = -1;
    if(cacheData1Len > dst*2+2)
    {
        endidx = cacheData1[dst*2+1];
        nextidx = cacheData1[dst*2+2];
    }
    else
    {
        nextidx = graphEdgeNUM;
    }

    int j = endidx;
    for(int i = lowerBound;i < upperBound;i+=2)
    {
        int src = edges[i];
        if(j < nextidx)
        {
            if(j < cacheData0Len)
                cacheData0[j] = src;
            j++;
        }
    }
}

__global__ void func2(int* cacheData0,
                    int* cacheData1,
                    const int* edges,
                    const int* bound,
                    const int cacheData0Len,
                    const int cacheData1Len,
                    const int edgesLen,
                    const int boundLen,
                    const int graphEdgeNUM)
{
    if(blockIdx.x >= (boundLen-1))
        return;
    
    const int lowerBound = bound[blockIdx.x];
    const int upperBound = bound[blockIdx.x+1];
    const int dst = edges[lowerBound+1];
    int endidx = -1;
    int nextidx = -1;
    if(cacheData1Len > dst*2+2)
    {
        endidx = cacheData1[dst*2+1];
        nextidx = cacheData1[dst*2+2];
    }
    else
    {
        nextidx = graphEdgeNUM;
    }

    // if(((lowerBound+2*threadIdx.x)<upperBound) && (endidx+threadIdx.x < nextidx) && (endidx+threadIdx.x < cacheData0Len))
    //     cacheData0[endidx+threadIdx.x] = edges[lowerBound+2*threadIdx.x];
    

    for(int i = threadIdx.x;(lowerBound+2*i)<upperBound;i+=1024)
        if((endidx+i) < nextidx && (endidx+i) < cacheData0Len)
            cacheData0[endidx+i] = edges[lowerBound+2*i];
}

void lanch_loading_halo(int* cacheData0,
                        int* cacheData1,
                        const int* edges,
                        const int* bound,
                        const int cacheData0Len,
                        const int cacheData1Len,
                        const int edgesLen,
                        const int boundLen,
                        const int graphEdgeNUM,
                        const int gpuDeviceIndex)
{   
    /* 指定使用的GPU序号 [0,torch.cuda.device_count()) */
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        //printf("No GPU devices found.\n");
        return;
    }
    else if(gpuDeviceIndex >= deviceCount || gpuDeviceIndex < 0){
        //printf("Wrong GPU Device Index:%d , Select Default Device Index:0 cuda:0.\n",gpuDeviceIndex);
        hipSetDevice(0);
    }
    else{
        //printf("Select GPU Device Index:%d , Please Prepare Pytorch Data tensor.to(device='cuda:%d')\n",gpuDeviceIndex,gpuDeviceIndex);
        hipSetDevice(gpuDeviceIndex);
    }

    dim3 grid((boundLen+1023)/1024);
    dim3 block(1024);
    func2<<<grid,block>>>(cacheData0,
                        cacheData1,
                        edges,
                        bound,
                        cacheData0Len,
                        cacheData1Len,
                        edgesLen,
                        boundLen,
                        graphEdgeNUM);
    
}

void lanch_loading_halo0(int* cacheData0,
                        int* cacheData1,
                        const int* edges,
                        const int cacheData0Len,
                        const int cacheData1Len,
                        const int edgesLen,
                        const int graphEdgeNUM,
                        const int gpuDeviceIndex)
{   
    /* 指定使用的GPU序号 [0,torch.cuda.device_count()) */
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        //printf("No GPU devices found.\n");
        return;
    }
    else if(gpuDeviceIndex >= deviceCount || gpuDeviceIndex < 0){
        //printf("Wrong GPU Device Index:%d , Select Default Device Index:0 cuda:0.\n",gpuDeviceIndex);
        hipSetDevice(0);
    }
    else{
        //printf("Select GPU Device Index:%d , Please Prepare Pytorch Data tensor.to(device='cuda:%d')\n",gpuDeviceIndex,gpuDeviceIndex);
        hipSetDevice(gpuDeviceIndex);
    }

    dim3 grid(1);
    dim3 block(1);
    
    func0<<<grid,block>>>(cacheData0,
                        cacheData1,
                        edges,
                        cacheData0Len,
                        cacheData1Len,
                        edgesLen,
                        graphEdgeNUM);
}